#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>  
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include<time.h>

#ifdef __unix
#define fopen_s(pFile,filename,mode) ((*(pFile))=fopen((filename),(mode)))==NULL
#endif

// 读取csv文件，获取数据测试数据
/* //for window run 
void read_csv(int file_no, float* M) {
    FILE* fp = NULL;
    errno_t err;
    char* word;
    char buffer[1000000];
    char* ptr = NULL;
    int i = 0;
    if (file_no == 0) {
        err = fopen_s(&fp, "../input/shape_1000/A_1000.csv", "r");
    }
    else if (file_no == 1) {
        err = fopen_s(&fp, "../input/shape_1000/B_1000.csv", "r");
    }
    else if (file_no == 2) {
        err = fopen_s(&fp, "../input/shape_1000/C_1000.csv", "r");
    }
    else {
        assert(0);
    }
    if (err) {
        assert(0);
    }
    while (fgets(buffer, sizeof(buffer), fp) != NULL) {
        word = strtok_s(buffer, ",", &ptr);
        while (word != NULL) {
            M[i++] = strtod(word, NULL);
            word = strtok_r(NULL, ",", &ptr);
        }
    }
}
*/
// for linux run
void read_csv(int file_no, float* array) {
    FILE * fp = NULL;
    char *line, *word;
    char buffer[100000];
    int i = 0;
    if(file_no == 0){
        printf("hello");
        fp = fopen("./input/shape_1000/A_1000.csv", "r");
    } else if(file_no == 1) {
        fp = fopen("./input/shape_1000/B_1000.csv", "r");
    } else if(file_no == 2) {
        fp = fopen("./input/shape_1000/C_1000.csv", "r");
    } else{
        assert(0);
    }
    if(fp == NULL){
        //assert(0);
    }
    while((line = fgets(buffer, sizeof(buffer), fp))!=NULL){
        word = strtok(line, ",");
        while (word != NULL){
            array[i++] = strtol(word, NULL, 10);
            word = strtok(NULL, ",");
        }
    }
}

// 核函数
__global__ void matrixMul(float* A, float* B, float* C, int m, int n, int p)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (row < m && col < p)
    {
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * p + col];
        }
        C[row * p + col] = sum;
    }
}

// 预热函数，come form the book <Professional CUDA C编程>
__global__ void warm_up_gpu() {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}
int main()
{
    int m, n, p;
    float* A, * B, * C,*Ct;
    int sizeA, sizeB, sizeC;
    // creat cuda timer
    hipEvent_t start, stop;
    float esp_time_gpu;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // creat cpu timer
    float esp_time_cpu_Data_Pre;
    float esp_time_cpu_Data_HostToDevice;
    float esp_time_cpu_Data_DeviceToHost;
    clock_t start_cpu, stop_cpu;


    start_cpu = clock();// start timing
    // Initialize matrix sizes
    m = n = p = 1000;
    sizeA = m * n;
    sizeB = n * p;
    sizeC = m * p;
    // Allocate host memory
    A = (float*)malloc(sizeA * sizeof(float));
    B = (float*)malloc(sizeB * sizeof(float));
    C = (float*)malloc(sizeC * sizeof(float));
    Ct = (float*)malloc(sizeC * sizeof(float));
    read_csv(0, A);
    read_csv(1, B);
    stop_cpu = clock();// end timing
    esp_time_cpu_Data_Pre = (float)(stop_cpu - start_cpu) / CLOCKS_PER_SEC * 1000 * 1000;


    read_csv(2, Ct);
    for(int i = 0; i < sizeC; i++)C[i] = 0.0f;
    // Allocate device memory
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, sizeA * sizeof(float));
    hipMalloc((void**)&d_B, sizeB * sizeof(float));
    hipMalloc((void**)&d_C, sizeC * sizeof(float));


    start_cpu = clock();// start timing
    // Copy host memory to device
    hipMemcpy(d_A, A, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB * sizeof(float), hipMemcpyHostToDevice);
    stop_cpu = clock();// end timing
    esp_time_cpu_Data_HostToDevice = (float)(stop_cpu - start_cpu)/ CLOCKS_PER_SEC*1000 * 1000;

    // Calculate grid size
    dim3 blockSize(32, 32);
    dim3 gridSize((m + blockSize.x - 1) / blockSize.x,
        (p + blockSize.y - 1) / blockSize.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);

    warm_up_gpu << <gridSize, blockSize>> > ();
    hipEventRecord(start, 0);// start
    // Launch kernel
    for (int i = 0; i < 10; i++) {
        matrixMul <<<gridSize, blockSize >>> (d_A, d_B, d_C, m, n, p);
    }
    hipEventRecord(stop, 0);// stop
    // device synchronize
    hipEventSynchronize(stop);
    hipEventElapsedTime(&esp_time_gpu, start, stop);


    start_cpu = clock();// start timing
    // Copy device memory to host
    hipMemcpy(C, d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);
    stop_cpu = clock();// end timing
    esp_time_cpu_Data_DeviceToHost = (float)(stop_cpu - start_cpu) / CLOCKS_PER_SEC * 1000*1000;
    


     //检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < sizeC; ++i) {
        maxError = fmax(maxError, fabs(C[i] - Ct[i]));
    }
    printf("矩阵大小:%d×%d\n", m, p);
    printf("最大误差:%f\n", maxError);

    printf("Time for the kernel: %f ms\n", esp_time_gpu/10);
    printf("Time for preparing data: %f us\n", esp_time_cpu_Data_Pre);
    printf("Time for Host to Device: %f us\n", esp_time_cpu_Data_HostToDevice);
    printf("Time for Device to Host: %f us\n", esp_time_cpu_Data_DeviceToHost);

 
    char filename[50];
    sprintf(filename, "output/time/time_cuda_%d.txt", m);
    FILE *file = fopen(filename, "w");
    if (file == NULL) {
        printf("无法打开文件.\n");
        return -1;
    }
    fprintf(file, "%f",  esp_time_gpu/10);
    fclose(file);
    
    sprintf(filename, "output/result/baseline_%d.txt", m);
    FILE *file1 = fopen(filename, "w");
    if (file1 == NULL) {
        printf("无法打开文件.\n");
        return -1;
    }
    for(int i=0 ;i<sizeC;i++){
        fprintf(file1,"%f\n",C[i]);
    }
    fclose(file1);
    
    sprintf(filename, "output/result/result_%d.txt", m);
    FILE *file2 = fopen(filename, "w");
    if (file2 == NULL) {
        printf("无法打开文件.\n");
        return -1;
    }
    for(int i=0 ;i<sizeC;i++){
        fprintf(file2,"%f\n", Ct[i]);
    }
    fclose(file2);
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Free host memory
    free(A);
    free(B);
    free(C);
    free(Ct);
    return 0;
}
