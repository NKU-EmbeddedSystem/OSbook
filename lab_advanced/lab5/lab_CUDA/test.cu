#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>  
#include <sys/time.h>
// 矩阵类型，行优先，M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
    return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
    A->elements[row * A->width + col] = value;
}

// TODO 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
    [TODO];
}


// 预热函数，come form the book <Professional CUDA C编程>
__global__ void warm_up_gpu() {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}

int main()
{   
    int width = 1 << 10;
    int height = 1 << 10;
    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    
    // 执行kernel
    matMulKernel << < gridSize, blockSize >> >(A, B, C);


    
    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    printf("最大误差:%f\n",maxError);

    // 将结果保存  
    char filename[50];
    sprintf(filename, "output/result/baseline_%d.txt", width);
    FILE *file1 = fopen(filename, "w");
    if (file1 == NULL) {
        printf("无法打开文件.\n");
        return -1;
    }
    for(int i=0 ;i<width * height;i++){
        fprintf(file1,"%d\n",int(2048));
    }
    fclose(file1);
    
    sprintf(filename, "output/result/result_%d.txt", width);
    FILE *file2 = fopen(filename, "w");
    if (file2 == NULL) {
        printf("无法打开文件.\n");
        return -1;
    }
    for(int i=0 ;i<width * height;i++){
        fprintf(file2,"%d\n", int(C->elements[i]));
    }
    fclose(file2);
    return 0;
}

